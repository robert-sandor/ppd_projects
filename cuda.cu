#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <>

#include "hip/hip_runtime.h"

static const int NUM_THREADS = 20;
static const int COLOR_CHANGE_AMOUNT = 25;

__global__ void red_filter(char* imageData, int rows, int columns, int channels) {
    int thread_id = threadIdx.x;
    int from = rows / NUM_THREADS * thread_id;
    int to = rows / NUM_THREADS * (thread_id + 1);

    for (int x = from; x < to; x++) {
        for (int y = 0; y < columns; y++) {
            imageData[x * columns * channels + y * channels] =
                    (char) (imageData[x * columns * channels + y * channels] - COLOR_CHANGE_AMOUNT < 0 ? 0 :
                                        imageData[x * columns * channels + y * channels] - COLOR_CHANGE_AMOUNT);
            imageData[x * columns * channels + y * channels + 1] =
                    (char) (imageData[x * columns * channels + y * channels + 1] - COLOR_CHANGE_AMOUNT < 0 ? 0 :
                                        imageData[x * columns * channels + y * channels + 1] - COLOR_CHANGE_AMOUNT);
            imageData[x * columns * channels + y * channels + 2] =
                    (char) (imageData[x * columns * channels + y * channels + 2] + COLOR_CHANGE_AMOUNT > 255 ? 0 :
                                        imageData[x * columns * channels + y * channels + 2] + COLOR_CHANGE_AMOUNT);
        }
    }
}

void image_cuda(char *imageData, size_t size, int rows, int cols, int channels) {
    char *dev_image;

    hipMalloc((void **) &dev_image, size);
    hipMemcpy(dev_image, imageData, size, hipMemcpyHostToDevice);

    red_filter << < 1, NUM_THREADS >> > (dev_image, rows, cols, channels);

    hipMemcpy(imageData, dev_image, size, hipMemcpyDeviceToHost);
    hipFree(dev_image);
}